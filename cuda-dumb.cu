#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdio.h>
#include <stdint.h>
#include <zlib.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <fcntl.h>
#include <limits.h>

typedef struct __ReadSeqList {
	char* sequence;
	unsigned length;
	struct __ReadSeqList* next;
} ReadSeqList;

typedef struct HashTable {
	unsigned int bits;
	unsigned int count;
	unsigned int read_count;
	unsigned long long int *keys;
    unsigned int *values;
} HashTable;



__device__ const unsigned char seq_nt4_table[256] = { // translate ACGT to 0123
	0, 1, 2, 3,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  3, 3, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  3, 3, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4
};


// funcion para calcular un hash de 64 bits
__device__ unsigned int hash_uint64(unsigned long long int key) {

	key = ~key + (key << 21);
	key = key ^ key >> 24;
	key = (key + (key << 3)) + (key << 8);
	key = key ^ key >> 14;
	key = (key + (key << 2)) + (key << 4);
	key = key ^ key >> 28;
	key = key + (key << 31);
	return (unsigned int)key;
}

HashTable* HashTable_init(unsigned int bits, unsigned int read_count){
    HashTable *ht;
    ht = (HashTable*)calloc(1, sizeof(HashTable));

	ht->read_count = read_count;
	ht->bits = bits;
	ht->count = 0;

    return ht;
}


void HashTable_destory(HashTable *ht) {
	if (!ht) return;
	free(ht);
}


__device__ unsigned int h2b(unsigned int hash, unsigned long long int bits) {
    return hash * 2654435769U >> (32 - bits);
}

__device__ void hash_insert(HashTable *ht, unsigned long long int kmer) {

	unsigned int iKey, last;
	//bool end = false;


	//iKey = last = hash_uint64(kmer) * (2654435769U >> (32 - ht->bits));
    iKey = last = h2b(hash_uint64(kmer), ht->bits);
    while (ht->values[iKey] > 0 && ht->keys[iKey] != kmer) {
        iKey = (iKey + 1U) & ((1U << ht->bits) - 1);
        if (iKey == last) break;
    }

    // Comprobar si se ha encontrado un slot vacío
    if (ht->values[iKey] == 0) { // no se ha encontrado la llave

        ht->keys[iKey] = kmer;
        ht->values[iKey] = 1;
        ++ht->count;

    } else {
        ht->values[iKey]++;
    } 

    /*
	while (true)
	{
		unsigned int prev = atomicCAS(ht->keys[iKey], NULL, iKey);

		if (prev == NULL || prev == kmer) {
			ht->keys[iKey] = kmer;
			atomicAdd(&(ht->values[iKey]), 1);

			if(prev == NULL) atomicAdd(&(ht->count), 1);

			return;
		}

		if(end) return;

		// Collition: Open addressing
		iKey = (iKey + 1U) & (ht->bits - 1);

		// loop back
		end = (iKey == last);

	}
    */

}

// insert k-mers in $seq to hash table $ht
__global__ void kernel_count_seq_kmers(HashTable *ht, int k, char **d_reads)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < ht->read_count) {

        int i, l;
		char *seq = d_reads[tid];
		//int len = strlen(seq);
		int len = 100;
        unsigned long long int x[2], mask = (1ULL<<k*2) - 1, shift = (k - 1) * 2;

		for (i = l = 0, x[0] = x[1] = 0; i < len; ++i) {
			int c = seq_nt4_table[(unsigned char)seq[i]];
			if (c < 4) { // not an "N" base
				x[0] = (x[0] << 2 | c) & mask;                  // forward strand
				x[1] = x[1] >> 2 | (unsigned long long int)(3 - c) << shift;  // reverse strand
				if (++l >= k) { // we find a k-mer

					unsigned long long int kmer = x[0] < x[1]? x[0] : x[1];
					hash_insert(ht, kmer); // only add one strand!
				}
			} else l = 0, x[0] = x[1] = 0; // if there is an "N", restart
		}
	}
}

__global__ void kernel_print_hist(const HashTable *ht, unsigned int *cnt_d)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned pos;


	if(threadIdx.x < ht->bits) {
		if (ht->values[tid] != 0) {
			pos = ht->values[tid] < 256 ? ht->values[tid] : 255;
            cnt_d[pos]++;
			//atomicAdd(&cnt_d[pos], (unsigned int)1);
		}
	}
}

static int count_file(const char *fn, int k, unsigned int p)
{
	//gzFile fp;
	//kseq_t *ks;
	HashTable *ht;
    unsigned int capacity = 1U << p;
    unsigned int cnt[256];
    unsigned int read_count = 0;




    // variables para cuda
	HashTable *ht_d;
	char **reads_d;
	unsigned int *cnt_d;

    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;

    fp = fopen(fn, "r");
    if (fp == NULL) exit(EXIT_FAILURE);

   
	ReadSeqList *current, *head;
	head = current = NULL;

    while ((read = getline(&line, &len, fp)) != -1) {

        read_count++;

		ReadSeqList *node = (ReadSeqList*)malloc(sizeof(ReadSeqList));
        node->sequence = (char*)malloc(strlen(line));
        strcpy(node->sequence, line);
        node->length = read;
        node->next =NULL;

        if(head == NULL){
            current = head = node;
        } else {
            current = current->next = node;
        }


    }

    fclose(fp);
    if (line) free(line);

    printf("%d\n", read_count);
    unsigned int i;

    char **reads = (char**)malloc(read_count * sizeof(char*)); 

	for(i=0, current = head; current; current=current->next){
        reads[i] = (char*)malloc(current->length);
        sprintf(reads[i], "%s", current->sequence);
        i++;
    }
   

    // inicializar hashtable
	ht = HashTable_init(p, read_count);

    
    unsigned long long int *keys_d;
    unsigned int *values_d;


	// allocate memory in device
	hipMalloc((void **)&ht_d, sizeof(HashTable));
   	hipMalloc((void **)&keys_d, capacity * sizeof(unsigned long long int));
	hipMalloc((void **)&values_d, capacity * sizeof(unsigned int));
	hipMalloc((void **)&cnt_d, 256 * sizeof(unsigned int));
   	hipMemset(keys_d, 0, capacity * sizeof(unsigned long long int));
	hipMemset(values_d, 0, capacity * sizeof(unsigned int));
	hipMemset(cnt_d, 0, 256 * sizeof(unsigned int));

    
	// copy data to device

    ht->keys = keys_d;
    ht->values = values_d;

	hipMemcpy(ht_d, ht, sizeof(HashTable), hipMemcpyHostToDevice);

	char **tmp = (char**)malloc (read_count * sizeof (char*));
    for (int i = 0; i < read_count; i++) {
        hipMalloc ((void **)&tmp[i], head->length * sizeof (char));
    }

	hipMalloc((void **)&reads_d, read_count * sizeof(char*));

    hipMemcpy(reads_d, tmp, read_count * sizeof (char*), hipMemcpyHostToDevice);
    for (int i = 0; i < read_count; i++) {
        hipMemcpy(tmp[i], reads[i], head->length * sizeof (char), hipMemcpyHostToDevice);
    }
    free(tmp);



    printf("total reads: %d\n", read_count);


    // invocar kernels
    int thr = 1024;

	//kernel_count_seq_kmers<<<1,1>>>(ht_d, k, reads_d);
	kernel_count_seq_kmers<<<ceil(read_count/thr), thr>>>(ht_d, k, reads_d);

    hipDeviceSynchronize();


	//kernel_print_hist<<<ceil(capacity/thr), thr>>>(ht_d, cnt_d);

    //cudaDeviceSynchronize();

	hipMemcpy(ht, ht_d, sizeof(HashTable), hipMemcpyDeviceToHost);
	hipMemcpy(ht->keys, keys_d, capacity * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	hipMemcpy(ht->values, values_d, capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(cnt, cnt_d, capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);


    printf("COUNT: %d\n\n", ht->count);

    // for(i = 0; i< 5000 ; i++){
    //     printf("key: ");
    //     printf("%"PRIu64"\t", ht->keys[i]);
    //     printf("value: %d\t", ht->values[i]);
    //     printf("collitions: %d\n", ht->collition[i]);
    // }

	for (i = 1; i < 256; ++i)
		printf("%d\t%ld\n", i, (long)cnt[i]);


	// limpieza
    hipFree(reads_d);
	hipFree(ht_d);
	hipFree(cnt_d);
	hipFree(keys_d);
	hipFree(values_d);
    return 0;
	// limpieza
    i = 0;
	for(current = head; current; current=current->next){
        free(current->sequence);
        free(current);
        free(reads[i]);
        i++;
    }

    free(reads);
	HashTable_destory(ht);
    return 0;
}


int main(int argc, char *argv[])
{
	int k = 31;
    unsigned int p = 27;

/*
	ketopt_t o = KETOPT_INIT;
	while ((c = ketopt(&o, argc, argv, 1, "k:", 0)) >= 0)
		if (c == 'k') k = atoi(o.arg);
	if (argc - o.ind < 1) {
		fprintf(stderr, "Usage: kc-c1 [-k %d] <in.fa>\n", k);
		return 1;
	}
*/
    k = (int)strtol(argv[1], NULL, 10);
	count_file(argv[2], k, p);

	return 0;
}
