#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdio.h>
#include <stdint.h>
#include <zlib.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <fcntl.h>
#include <limits.h>

#define __STDC_FORMAT_MACROS
#include <inttypes.h>

#include "kseq.h" // FASTA/Q parser
//extern "C"
//{
	//#include "kseq.h" // FASTA/Q parser
	//#include "ketopt.h" // command-line argument parser
//}

KSEQ_INIT(gzFile, gzread)


__device__ uint32_t d_capacity;


typedef struct __ReadSeqList {
	char* sequence;
	unsigned length;
	struct __ReadSeqList* next;
} ReadSeqList;

typedef struct HashTable {
	uint32_t bits;
	uint32_t count;
	//uint32_t *collition;
	uint64_t *keys;
    uint32_t *values;
} HashTable;



const unsigned char seq_nt4_table[256] = { // translate ACGT to 0123
	0, 1, 2, 3,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  3, 3, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  3, 3, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4
};


// funcion para calcular un hash de 64 bits
__device__ uint32_t hash_uint64(uint64_t key) {

	key = ~key + (key << 21);
	key = key ^ key >> 24;
	key = (key + (key << 3)) + (key << 8);
	key = key ^ key >> 14;
	key = (key + (key << 2)) + (key << 4);
	key = key ^ key >> 28;
	key = key + (key << 31);
	return (uint32_t)key;
}

void HashTable_init(HashTable *ht, uint32_t bits){
	uint32_t capacity = 1U << bits;
	ht->bits = capacity;
	ht->count = 0;
}


void HashTable_destory(HashTable *ht) {
	if (!ht) return;
	free((void *)ht->keys);
	free((void *)ht->values);
	free(ht);
}


__device__ void hash_insert(HashTable *ht, uint64_t kmer) {

	unsigned int iKey, last;
	bool end = false;

	iKey = last = hash_uint64(kmer) * (2654435769U >> (32 - ht->bits));

	while (true)
	{
		uint32_t prev = atomicCAS(&ht.keys[iKey], NULL, iKey);

		if (prev == NULL || prev == kmer) {
			ht->keys[iKey] = kmer;
			atomicAdd(&(ht->values[iKey]), 1);

			if(prev == NULL) atomicAdd(&(ht->count), 1);

			return;
		}

		if(end) return;

		// Collition: Open addressing
		iKey = (iKey + 1U) & (d_capacity - 1);

		// loop back
		end = (iKey == last);

	}

}

// insert k-mers in $seq to hash table $ht
__global__ void kernel_count_seq_kmers(HashTable *ht, int k, char **d_reads, uint32_t read_count)
{
	if(threadIdx.x < read_count) {

        int i, l;
		char *seq = d_reads[threadIdx.x];
		int len = strlen(seq);
        uint64_t x[2], mask = (1ULL<<k*2) - 1, shift = (k - 1) * 2;

		for (i = l = 0, x[0] = x[1] = 0; i < len; ++i) {
			int c = seq_nt4_table[(uint8_t)seq[i]];
			if (c < 4) { // not an "N" base
				x[0] = (x[0] << 2 | c) & mask;                  // forward strand
				x[1] = x[1] >> 2 | (uint64_t)(3 - c) << shift;  // reverse strand
				if (++l >= k) { // we find a k-mer

					uint64_t kmer = x[0] < x[1]? x[0] : x[1];
					hash_insert(ht, kmer); // only add one strand!

				}
			} else l = 0, x[0] = x[1] = 0; // if there is an "N", restart
		}
	}
}

__global__ void kernel_print_hist(const HashTable *ht, uint64_t *cnt_d)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned pos;


	if(threadIdx.x < 256) {
		if (ht->values[tid] != 0) {
			pos = ht->values[tid] < 256 ? ht->values[tid] : 255;
			atomicAdd(&cnt_d[pos], (uint64_t)1);
		}
	}
}

static void count_file(const char *fn, int k, uint32_t p)
{
	gzFile fp;
	kseq_t *ks;
	HashTable *ht;

	if ((fp = gzopen(fn, "r")) == 0) return 0;
	ks = kseq_init(fp); // descriptor fichero fastaq


	ReadSeqList *current, *head;
	head = current = NULL;

	// leer los datos del fichero de entrada y guardarlos en memoria
    uint32_t read_count = 0;
	while (kseq_read(ks) >= 0) {
        read_count++;

		ReadSeqList *node = (ReadSeqList*)malloc(sizeof(ReadSeqList));
        node->sequence = (char*)malloc(strlen(ks->seq.s) + 1);
        strcpy(node->sequence, ks->seq.s);
        node->length = ks->seq.l;
        node->next =NULL;

        if(head == NULL){
            current = head = node;
        } else {
            current = current->next = node;
        }

	}

	kseq_destroy(ks);
	gzclose(fp);

    unsigned int i;
    // crear un array de tamaño fijo para almacenar las lecturas
    // char **reads = malloc(read_count * sizeof(char*));



	// variables para cuda
	HashTable *ht_d;
	char **d_reads;
	uint64_t *cnt_d;

	// inicializar hashtable
	HashTable_init(ht, p);

	uint32_t capacity = 1U << p;





	// allocate memory in device
	hipMalloc((void **)&d_reads, read_count * sizeof(char *));
	hipMalloc((void **)&ht_d, sizeof(HashTable));
	hipMalloc((void **)ht_d->keys, capacity * sizeof(uint64_t));
	hipMalloc((void **)ht_d->values, capacity * sizeof(uint32_t));
	//hipMalloc((void **)ht_d->collition, capacity * sizeof(uint32_t));
	hipMalloc((void **)&cnt_d, 256 * sizeof(uint64_t));

	hipMemset(ht_d->keys, 0, capacity * sizeof(uint64_t));
	hipMemset(ht_d->values, 0, capacity * sizeof(uint32_t));
	hipMemset(cnt_d, 0, 256 * sizeof(uint64_t));

	// copy data to device
	hipMemcpy(ht_d, ht, sizeof(HashTable), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(&d_capacity), capacity, sizeof(uint32_t));





	// copiar los read a la memoria de la GPU
	char **d_temp_reads = (char **)malloc(read_count * sizeof(char *));

	for(i=0, current = head; current; current=current->next){
		hipMalloc((void **)&(d_temp_reads[i]), strlen(current->sequence)  * sizeof(char));
		hipMemcpy(d_temp_reads[i], current->sequence, strlen(current->sequence) * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(d_reads + i, &(d_temp_reads[i]), sizeof(char *), hipMemcpyHostToDevice);
		i++;
    }



    //printf("%s\n", reads[0]);

    printf("total reads: %d\n", read_count);



	kernel_count_seq_kmers<<<ceil(read_count/1024), 1024>>>(ht_d, k, d_reads, read_count);

	kernel_print_hist<<<ceil(ht_d->count/256), 256>>>(ht_d, cnt_d);



	hipMemcpy(ht, ht_d, sizeof(HashTable), hipMemcpyDeviceToHost);
	hipMemcpy(ht->keys, ht_d->keys, capacity * sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipMemcpy(ht->values, ht_d->values, capacity * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(cnt, cnt_d, capacity * sizeof(uint64_t), hipMemcpyDeviceToHost);


    printf("COUNT: %d\n\n", ht->count);

    // for(i = 0; i< 5000 ; i++){
    //     printf("key: ");
    //     printf("%"PRIu64"\t", ht->keys[i]);
    //     printf("value: %d\t", ht->values[i]);
    //     printf("collitions: %d\n", ht->collition[i]);
    // }

	for (i = 1; i < 256; ++i)
		printf("%d\t%ld\n", i, (long)cnt[i]);


	// limpieza
	for(current = head; current; current=current->next){
        free(current->sequence);
        free(current);
    }

	hipFree(d_reads);
	hipFree(ht_d);
	hipFree(cnt_d);

	HashTable_destory(ht);
}


int main(int argc, char *argv[])
{
	HashTable *ht;
	int c, k = 31;
    uint32_t p = 27;
    /*
	ketopt_t o = KETOPT_INIT;
	while ((c = ketopt(&o, argc, argv, 1, "k:", 0)) >= 0)
		if (c == 'k') k = atoi(o.arg);
	if (argc - o.ind < 1) {
		fprintf(stderr, "Usage: kc-c1 [-k %d] <in.fa>\n", k);
		return 1;
	}*/

	//count_file(argv[o.ind], k, p);
	count_file("../dataset/M_abscessus_HiSeq_10M.fa.gz", k, p);

	return 0;
}
